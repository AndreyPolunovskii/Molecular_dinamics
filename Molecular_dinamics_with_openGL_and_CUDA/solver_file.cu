#include "hip/hip_runtime.h"
#include "include_files.h"
#include "parameters.h"
#include "vars.h"

#include "solver_file.h"
#include "source_funct.cuh"
#include "add_functions.h"

//---------------------------------------------------
int init_function(void)
{
	T = col_steps * ht;
	
	time1 = clock();
	
	std::cout << " Длина отрезка интегрирования по времени = " << col_steps * ht << std::endl;

      
    //Сколько устройств CUDA установлено на PC.
    hipGetDeviceCount(&deviceCount);
    printf("Device count: %d\n\n", deviceCount);


    pow_sigma_12 = 12.f*pow(sigma,12);
	pow_sigma_6 = 6.f*pow(sigma,6);
	ht_ht = ht*ht;

    // размер массива из N 3dвекторов
	size = N * sizeof(vector3d);

    // выделяем память на видюхе
	hipMalloc( (void**)&dev_r0, size );
	hipMalloc( (void**)&dev_v0, size );
	hipMalloc( (void**)&dev_a0, size );
	        
	 hipMalloc( (void**)&dev_r1, size );
	 hipMalloc( (void**)&dev_v1, size );
	 hipMalloc( (void**)&dev_a1, size );

	       
	 // выделяем память в оперативке 	        
	 r0 = (vector3d*) malloc( size );
	 v0 = (vector3d*) malloc( size );
	 a0 = (vector3d*) malloc( size );
	        
	 r1 = (vector3d*) malloc( size );
	 v1 = (vector3d*) malloc( size );
	 a1 = (vector3d*) malloc( size );
	        
	       // задаем н. у.
	       // нельзя задавать частицы друг в друге!!!

				 // задаем вдоль линии x частицы
               /*  r0[l].y =  (1./2. - alfa )* L + ( ((scalar) l)/((scalar)N))*( 2*alfa )*L;//l*hx;
                 r0[l].x =  ((scalar) ((1./2. - alfa )*L+ rand() % (int)(2*alfa*L) ));//l*hx+1;
                 r0[l].z =  (1./2. - alfa )* L + ( ((scalar) l)/((scalar)N))*( 2*alfa )*L;//l*hx+2;*/
               
                for (int i=0 ; i < Nx; ++i)
                 for (int j=0 ; j < Nx; ++j)
                  for (int k=0 ; k < Nx; ++k)
	          {  
                 int iter = i + j*Nx + k *Nx*Nx;       
                 r0[iter].y =  (1./2.-alfa)*L + ( (scalar) i)/((scalar) Nx )*(2.*alfa*L);
                 r0[iter].x =  (1./2.-alfa)*L + ( (scalar) j)/((scalar) Nx )*(2.*alfa*L);
                 r0[iter].z =  (1./2.-alfa)*L + ( (scalar) k)/((scalar) Nx )*(2.*alfa*L);
               }  
          
                
              for (int l=0 ; l < N; ++l)
	        { 
                 v0[l].x = 0;// (scalar) (-MAX_V + rand() % (int)2*MAX_V)/((scalar)DEV_V);
                 v0[l].y = 0;// (scalar) (-MAX_V + rand() % (int)2*MAX_V)/((scalar)DEV_V);
                 v0[l].z = 0;// (scalar) (-MAX_V + rand() % (int)2*MAX_V)/((scalar)DEV_V);
                 
                 
             }
             
        /*     	        for (ssize_t l=0 ; l < N; ++l)
	        {
				std::cout << r0[l].x << std::endl;
                std::cout << r0[l].y << std::endl;
                std::cout << r0[l].z << std::endl;
			} */
       
        // задаем размерности сетки на видеокарте         
        dim3 threads ( BLOCK_SIZE );
        dim3 blocks  ( N / threads.x);
       
       // копируем содержимое переменных из оперативы в переменные на девайсе      
       hipMemcpy(dev_r0 , r0, size, hipMemcpyHostToDevice );
       
       acceleration<<< blocks , threads >>>(dev_a0,dev_r0,sigma,pow_sigma_12,pow_sigma_6,eps,inv_m,(int)N);
    
       hipMemcpy(a0 , dev_a0, size, hipMemcpyDeviceToHost );
   
     flag = 0;
    
     outdata.open(file_str1);
    
     return 0;
	        
}

//--------------------------------------------------

int main_solver(int k) {

           // задаем размерности сетки на видеокарте         
           dim3 threads ( BLOCK_SIZE );
           dim3 blocks  ( N / threads.x);
            	       
           // запускаем цикл по времени  
         //---------------------------------------------------------
		   time2 = clock();
		   
		     // create cuda event handles
            hipEvent_t start, stop;
            float gpuTime = 0.0f;
            hipEventCreate ( &start );
            hipEventCreate ( &stop );
            
             // asynchronously issue work to the GPU (all to stream 0)
            hipEventRecord ( start, 0 );
		   
	       // копируем содержимое переменных из оперативы в переменные на девайсе      
	       hipMemcpy(dev_r0 , r0, size, hipMemcpyHostToDevice );
	       hipMemcpy(dev_v0 , v0, size, hipMemcpyHostToDevice );
	       hipMemcpy(dev_a0 , a0, size, hipMemcpyHostToDevice );
	        	    
	       // алгоритм Верле    
	       coordinates<<< blocks , threads >>>(dev_r1,dev_r0,dev_v0,dev_a0,ht,ht_ht);
	       acceleration<<< blocks , threads >>>(dev_a1,dev_r1,sigma,pow_sigma_12,pow_sigma_6,eps,inv_m,(int)N);
	       velocity<<< blocks , threads >>>(dev_v1,dev_v0,dev_a0,dev_a1,ht);

	        // Обратно копируем       
	       hipMemcpy(r1 , dev_r1, size, hipMemcpyDeviceToHost );
	       hipMemcpy(v1 , dev_v1, size, hipMemcpyDeviceToHost );
	       hipMemcpy(a1 , dev_a1, size, hipMemcpyDeviceToHost );
	       
	       //определяем точку завершения работы видеокарты
	       hipEventRecord ( stop, 0 );
           hipEventSynchronize ( stop );
           hipEventElapsedTime ( &gpuTime, start, stop );
	       
	       #pragma parallel for
	       for (ssize_t i=0 ; i < N; ++i)
	       {
                 r0[i].x = r1[i].x ;
                 r0[i].y = r1[i].y ;
                 r0[i].z = r1[i].z ;
                 
                 
                 
                 v0[i].x = v1[i].x ;
                 v0[i].y = v1[i].y ;
                 v0[i].z = v1[i].z ;
                 
    
                 
                 a0[i].x = a1[i].x ;
                 a0[i].y = a1[i].y ;
                 a0[i].z = a1[i].z ;

                 if (initcond == 1) // отражающие граничные условия
                {
					
                 if ( r0[i].x < 0)
                 {
                       r0[i].x = -r0[i].x;
                       v0[i].x = -v0[i].x;
                       a0[i].x = -a0[i].x;
                 }
                  
                 if ( r0[i].y < 0)
                 {
                       r0[i].y = -r0[i].y;
                       v0[i].y = -v0[i].y;
                       a0[i].y = -a0[i].y; 
                 }
                       
                 if ( r0[i].z < 0)
                 {
                       r0[i].z = -r0[i].z;
                       v0[i].z = -v0[i].z;
                       a0[i].z = -a0[i].z;
                 }
                       
                 if ( r0[i].x > L)
                 {
                       r0[i].x = 2*L - r0[i].x ;
                       v0[i].x =  - v0[i].x ;
                       a0[i].x =  - a0[i].x ;
                 }
                  
                 if ( r0[i].y > L)
                 {
                       r0[i].y = 2*L - r0[i].y ;
                       v0[i].y = - v0[i].y ;
                       a0[i].y = - a0[i].y ; 
				 }
                       
                 if ( r0[i].z > L)
                 {
                      r0[i].z = 2*L - r0[i].z ;
                      v0[i].z = - v0[i].z ;
                      a0[i].z = - a0[i].z ;
                 }
                 
                }
                else if (initcond == 2) // периодические граничные условия
                {
					
                 if ( r0[i].x < 0)
                       r0[i].x = L+r0[i].x;
            
                  
                 if ( r0[i].y < 0)
                       r0[i].y = L+r0[i].y;

                       
                 if ( r0[i].z < 0)
                       r0[i].z = L+r0[i].z;

                       
                 if ( r0[i].x > L)
                       r0[i].x =  r0[i].x - L ;

                  
                 if ( r0[i].y > L)
                       r0[i].y =  r0[i].y - L ;
                       
                 if ( r0[i].z > L)
                      r0[i].z =  r0[i].z - L ;
                 
                }

	       }
	       
	       


           
	                          
	       // сохраняем координаты в файл
           outdata << k*ht<<" ";
           for (int j=0 ; j < N; ++j)
           {
               outdata <<  r0[j].x << " " ;
               outdata <<  r0[j].y << " " ;
               outdata <<  r0[j].z << " " ;
            }
            
            outdata << std::endl; 
           
           #pragma parallel for
           for (int j=0 ; j < N; ++j)
           {
             // соответственно рисуем частички
               glPushMatrix();
               glTranslatef( (r0[j].x-L/2)*2/L, (r0[j].y-L/2)*2/L, (r0[j].z-L/2)*2/L );
               glColor3f(0.82, 0.25, 0.078);
               glutSolidSphere(r, 20, 10);
               glPopMatrix();   
           }




              time1 = clock() - time1;
              time2 = clock() - time2;
             
             if (k % 100 == 0)
            { 
			  printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime );
		      std::cout << k <<"-ый шаг интегрирования" << std::endl;
              std::cout <<"x координата " << (int)N/2 <<"-ой частицы: "<< r0[(int)N/2].x << " в " << k*ht <<"-ую расчетную секунду" << std::endl;
              std::cout <<"скорость " << (int)N/2 <<"-ой частицы: "<< sqrt( pow(v0[(int)N/2].x,2) + pow(v0[(int)N/2].y,2) + pow(v0[(int)N/2].z,2) ) << " в " << k*ht <<"-ую расчетную секунду" << std::endl;
              std::cout <<"средняя температура : " << find_temperature(v0,(int)N,m) << std::endl;
              std::cout <<"среднее давление : " << find_pressure(v0,(int)N,m) << std::endl;
              std::cout << "проинтегрирован отрезок = " << k*ht << std::endl;
              std::cout << "выполнено :" << k*ht*100/T <<"%" << std::endl;

              std::cout << "время, потраченное на одну итерацию :" << ((double) time2)/CLOCKS_PER_SEC << " секунд(ы)" << std::endl;
              std::cout << "общее прошедшее время : " << ((double) time1)/CLOCKS_PER_SEC << " секунд(ы)" << std::endl;
    
              std::cout << "----------------------------" << std::endl;
              std::cout << std::endl;
            }
            
            
	        	                   
            return 0;
}
  
//---------------------------------------------------
int free_function(void)
{
	// освобождаем память
    free( r0 );
    free( v0 );
    free( a0 );
	        
    free( r1 );
    free( v1 );
    free( a1 );
	        
    hipFree( dev_r0 );
    hipFree( dev_v0 );
    hipFree( dev_a0 );
	         
    hipFree( dev_r1 );
    hipFree( dev_v1 );
    hipFree( dev_a1 );
    
    outdata.close();
    
    return 0;
	        
}
